#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include "caffe/layers/kl_loss_layer.hpp"

namespace caffe {
template <typename Dtype>
__global__ void KlForward(const int n, const Dtype* in, const Dtype* alpha, Dtype* out){
// f(x) = e^(-alpha) * (x-1/2) + alpha/2    if |x| > 1
//      = e^(-alpha) * x^2 * 1/2 + alpha/2  if |x| <= 1
  CUDA_KERNEL_LOOP(index, n) {
    Dtype x = in[index];
    Dtype abs_x = abs(x);
    Dtype a = alpha[index];
    if (abs_x > 1) {
      out[index] = exp(-a) * (abs_x - 0.5) + a * 0.5;
    }
    else {
      out[index] = exp(-a) * x * x * 0.5 + a * 0.5; 
    }
  }
}


template <typename Dtype>
void KlLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  int count = bottom[0]->count();
  caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[2]->gpu_data(), diff_.mutable_gpu_data());
  KlForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, diff_.gpu_data(), bottom[1]->gpu_data(), error_.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;

  Dtype loss;
  caffe_gpu_dot(count, ones_.gpu_data(), error_.gpu_data(), &loss);
  top[0]->mutable_cpu_data()[0] = loss / count;
}

template <typename Dtype>
__global__ void KlBackward(const int n, const Dtype* in1, const Dtype* in2,
    const Dtype* in3, const Dtype* in4, Dtype* out1, Dtype* out2) {
// f'(xe) = e^(-alpha) * (xe - xg)  if |xg - xe| <= 1
//        = -e^(-alpha)      if |xg - xe| > 1 and xg > xe
//        = e^(-alpha)       if |xg - xe| > 1 and xg < xe
//
// f'(alpha) = -(xg - xe)^2 * 0.5 * e^(-alpha) + 0.5   if |xg - xe| <= 1 
//           = -(abs(xg-xe) - 0.5) * e^(-alpha) + 0.5
        
  CUDA_KERNEL_LOOP(index, n) {
    Dtype d = in1[index];//xe - xg
    Dtype xe = in2[index];
    Dtype xg = in3[index];
    Dtype alpha = in4[index];
    Dtype abs_d = abs(d);
    Dtype ea = exp(-alpha);
    if (abs_d <= 1) {
      out1[index] = ea * d;
      out2[index] = -d*d * 0.5 * ea + 0.5;
    }
    else {
      if (xg > xe) { 
        out1[index] = -ea;
      }
      else {
        out1[index] = ea;
      }
      out2[index] = -(abs_d - 0.5) * ea + 0.5;
    }
  }
}

template <typename Dtype>
void KlLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[2]){
    LOG(FATAL) << this->type() << "Layer cannot backpropage to gt input!";
  }
  if (propagate_down[0] && propagate_down[1]){
    int count = diff_.count();
    Dtype* bottom_diff1 = bottom[0]->mutable_gpu_diff();
    Dtype* bottom_diff2 = bottom[1]->mutable_gpu_diff();
    KlBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), bottom[0]->gpu_data(),bottom[2]->gpu_data(),
      bottom[1]->gpu_data(), bottom_diff1, bottom_diff2);
    CUDA_POST_KERNEL_CHECK;
    const  Dtype loss_weight = top[0]->cpu_diff()[0] / count;
    caffe_gpu_scal(count, loss_weight , bottom_diff1);
    caffe_gpu_scal(count, loss_weight , bottom_diff2);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KlLossLayer);

} // namespace caffe
